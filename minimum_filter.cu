#include "hip/hip_runtime.h"
__global__ void d_minfilter_x(float *src, float *dst, int width, int height, int r, int BLOCKSIZE){
	int i, j;
	int mask, len, extra, num, head, rear;
	int bid, tid;
	bid = blockIdx.x;
	tid = threadIdx.x;

	__shared__ float g[MAX_SIZE]; 
	__shared__ float h[MAX_SIZE];

	mask = 2 * r + 1;							
	len = width + 2 * r + mask - (width + 2 * r) % mask;	
	extra = len - width - r;						
	num = len / mask;							
	if (bid<height){

		for (i = tid; i<r; i += BLOCKSIZE){
			g[i] = MAX_VALUE;
			h[i] = MAX_VALUE;
		}
		__syncthreads();
		for (i = tid; i<width; i += BLOCKSIZE){
			g[r + i] = src[bid * width + i];	
			h[r + i] = src[bid * width + i];
		}
		__syncthreads();
		for (i = tid; i<extra; i += BLOCKSIZE){
			g[r + width + i] = MAX_VALUE;
			h[r + width + i] = MAX_VALUE;
		}
		__syncthreads();


		for (i = tid; i<num; i += BLOCKSIZE){
			head = i*mask;
			rear = (i + 1)*mask - 1;
			for (j = head + 1; j<(head + mask); j++){
				g[j] = (g[j - 1] < g[j]) ? g[j - 1] : g[j];
				h[rear - j + head] = (h[rear - j + head + 1] < h[rear - j + head]) ? h[rear - j + head + 1] : h[rear - j + head];
			}
		}
		__syncthreads();


		for (i = tid; i<width; i += BLOCKSIZE)
			dst[bid * width + i] = (g[i + r + r] > h[i]) ? h[i] : g[i + r + r];
	}
}